#include "hip/hip_runtime.h"
#include <stdint.h>
#include <sys/time.h>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

using namespace std;

#define MAX_DEPTH 5
#define MAX_FIELD_CHARS 200
#define MAX_FIELDS 10

#define NUM_SMS 110
// must be power of two
#define BLOCK_SIZE 256
// must be power of two
#define NUM_THREADS_PER_SM 2048
#define NUM_BLOCKS_PER_SM (NUM_THREADS_PER_SM / BLOCK_SIZE)
#define NUM_BLOCKS (NUM_SMS * NUM_BLOCKS_PER_SM)
#define NUM_THREADS (NUM_THREADS_PER_SM * NUM_SMS)

#define MIN(x, y) (((x) < (y)) ? (x) : (y))

typedef enum {EXP_KEY, IN_KEY, EXP_COL, EXP_VAL, IN_VAL, EXP_COM} state;

typedef uint8_t uint1_t;

typedef struct __attribute__((packed)) {
  uint8_t expected_match_state;
  uint8_t next_match_state;
  uint8_t expected_input;
} split_entry;

typedef struct __attribute__((packed)) {
  uint8_t next_match_state;
  uint8_t expected_input;
} seq_entry;

__global__ void run(uint8_t *input_full, uint8_t num_seq_confs, uint8_t num_split_confs, uint32_t input_count,
  uint8_t *output_full, uint32_t *output_count) {

  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint8_t *input_buf = input_full + index * input_count;
  uint8_t *output_buf = output_full + index * input_count;

  uint32_t input_idx = 0;
  uint32_t output_buf_idx = 0;
  state parse_state = EXP_VAL;
  uint8_t match_state = 0; // caps number of chars in fields to match at ~254
  uint1_t in_string_value = 0;
  uint8_t last_char = ' ';
  uint8_t nest_depth = 0;

  uint8_t state_stack[MAX_DEPTH];
  uint8_t stack_ptr = 0;
  seq_entry seq_trans[MAX_FIELD_CHARS];
  split_entry split_trans[MAX_FIELDS];

  #define IS_WHITESPACE(c) ((c) == ' ' || (c) == '\n' || (c) == '\t')
  #define POP_STATE_STACK do {\
            if (match_state == MAX_FIELD_CHARS) {\
              output[output_idx++] = ',';\
            }\
            match_state = state_stack[--stack_ptr];\
          } while (0)


  // match_states are exactly 8 bits only when MAX_FIELD_CHARS is >= 127
  for (uint8_t i = 0; i < num_seq_confs; i++) {
    seq_trans[i] = (seq_entry){input_buf[input_idx], input_buf[input_idx + 1]};
    input_idx += 2;
  }
  for (uint8_t i = 0; i < num_split_confs; i++) {
    split_trans[i] = (split_entry){input_buf[input_idx], input_buf[input_idx + 1], input_buf[input_idx + 2]};
    input_idx += 3;
  }
  for (uint8_t i = num_split_confs; i < MAX_FIELDS; i++) {
    split_trans[i] = (split_entry){0, 0, 0}; // need to zero-initialize since all entries checked below
  }
  #define BUF_SIZE 512
  uint8_t input[BUF_SIZE];
  uint8_t output[BUF_SIZE]; // it is possible that output is larger than input, but with fairly large BUF_SIZE
  // on the kafka_json dataset it shouldn't happen
  for (uint32_t ii = input_idx; ii < input_count; ii += BUF_SIZE) {
    for (uint32_t i = ii; i < MIN(input_count, ii + BUF_SIZE); i++) {
      input[i - ii] = input_buf[i];
    }
    uint32_t output_idx = 0;
    for (uint32_t i = 0; i < MIN(input_count - ii, BUF_SIZE); i++) {
      #define EMIT_CUR_TOKEN do {\
                if (match_state == MAX_FIELD_CHARS) {\
                  output[output_idx++] = input[i];\
                }\
              } while (0)

      // only need a next variable for parse_state because the other two state variables used in if conditions
      // (in_string_value and nest_depth) are not at risk of an erroneous read after write
      state next_parse_state = parse_state;
      if (parse_state == EXP_VAL) {
        if (input[i] == '{') {
          next_parse_state = EXP_KEY;
          nest_depth++;
        } else if (nest_depth != 0 && !IS_WHITESPACE(input[i])) {
          // at nest_depth of 0 we only accept new records
          EMIT_CUR_TOKEN;
          next_parse_state = IN_VAL;
          in_string_value = input[i] == '"';
        }
      }
      if (parse_state == IN_VAL) {
        if (in_string_value) {
          EMIT_CUR_TOKEN;
          if (input[i] == '"' && last_char != '\\') {
            in_string_value = 0;
          }
        } else if (input[i] != '}') {
          if (input[i] == ',') {
            next_parse_state = EXP_KEY;
            POP_STATE_STACK;
          } else {
            EMIT_CUR_TOKEN;
          }
        }
      }
      if (input[i] == ',' && parse_state == EXP_COM) {
        next_parse_state = EXP_KEY;
        POP_STATE_STACK;
      }
      if (input[i] == '}' &&
        (parse_state == EXP_KEY || parse_state == EXP_COM || (parse_state == IN_VAL && !in_string_value))) {
        if (parse_state == EXP_COM || parse_state == IN_VAL) {
          POP_STATE_STACK;
        }
        if (nest_depth == 1) {
          output[output_idx++] = '/'; // record separator
          next_parse_state = EXP_VAL;
        } else {
          next_parse_state = EXP_COM;
        }
        nest_depth--;
      }
      if (input[i] == '"' && parse_state == IN_KEY) {
        next_parse_state = EXP_COL;
      }
      if (input[i] == ':' && parse_state == EXP_COL) {
        next_parse_state = EXP_VAL;
      }
      uint1_t entering_key = input[i] == '"' && parse_state == EXP_KEY;
      if (entering_key) {
        next_parse_state = IN_KEY;
        state_stack[stack_ptr++] = match_state;
      }
      if ((parse_state == IN_KEY || entering_key) && match_state != MAX_FIELD_CHARS &&
        (match_state != 0 || nest_depth == 1)) {
        // only allow match to start at top level
        if (input[i] == seq_trans[match_state].expected_input) {
          if (seq_trans[match_state].next_match_state == MAX_FIELD_CHARS) {
            output[output_idx++] = match_state;
          }
          match_state = seq_trans[match_state].next_match_state;
        } else {
          uint8_t next_match_state = 0;
          for (uint8_t j = 0; j < MAX_FIELDS; j++) {
            if (match_state == split_trans[j].expected_match_state && input[i] == split_trans[j].expected_input) {
              if (split_trans[j].next_match_state == MAX_FIELD_CHARS) {
                output[output_idx++] = match_state;
              }
              next_match_state = split_trans[j].next_match_state;
            }
          }
          match_state = next_match_state;
        }
      }

      last_char = input[i];
      parse_state = next_parse_state;
    }
    for (uint32_t i = 0; i < output_idx; i++) {
      output_buf[output_buf_idx + i] = output[i];
    }
    output_buf_idx += output_idx;
  }
  *(output_count + index) = output_buf_idx;
}

int main(int argc, char **argv) {
  uint32_t GLOBAL_CHARS = 10000000;
  uint32_t CHARS = atoi(argv[1]);

  // extracts "ad_id" and "ad_type"
  uint8_t seq_confs[] = {1, 34, 2, 97, 3, 100, 4, 95, 5, 105, 6, 100, 200, 34, 8, 121, 9, 112, 10, 101, 200, 34};
  uint8_t split_confs[] = {4, 7, 116};

  ifstream infile("kafka-json.txt");
  string line;

  uint32_t conf_size = sizeof(seq_confs) + sizeof(split_confs);
  uint32_t input_buf_size = conf_size + GLOBAL_CHARS;
  uint8_t *input_buf = new uint8_t[input_buf_size];
  uint32_t global_chars = 0;
  uint64_t chars = 0;
  memcpy(input_buf + global_chars, seq_confs, sizeof(seq_confs));
  global_chars += sizeof(seq_confs);
  memcpy(input_buf + global_chars, split_confs, sizeof(split_confs));
  global_chars += sizeof(split_confs);
  while (getline(infile, line)) {
    if (chars == 0 && global_chars + line.length() > conf_size + CHARS) {
      chars = global_chars;
    }
    if (global_chars + line.length() > input_buf_size) {
      break;
    }
    memcpy(input_buf + global_chars, line.c_str(), line.length());
    global_chars += line.length();
  }

  uint8_t *combined_input = new uint8_t[chars * NUM_THREADS];
  for (uint64_t i = 0; i < NUM_THREADS; i++) {
    memcpy(combined_input + i * chars, input_buf, conf_size);
    memcpy(combined_input + i * chars + conf_size, input_buf + conf_size + i * 10, chars - conf_size);
  }

  uint8_t *output_buf = new uint8_t[chars];
  uint32_t output_count;

  uint8_t *output_dev, *input_dev;
  uint32_t *output_count_dev;
  hipSetDevice(0);
  hipMalloc((void **) &output_dev, chars * NUM_THREADS);
  hipMalloc((void **) &input_dev, chars * NUM_THREADS);
  hipMalloc((void **) &output_count_dev, sizeof(uint32_t) * NUM_THREADS);

  hipMemcpy(input_dev, combined_input, chars * NUM_THREADS, hipMemcpyHostToDevice);

  struct timeval start, end, diff;
  gettimeofday(&start, 0);
  run<<<NUM_BLOCKS, BLOCK_SIZE>>>(input_dev, sizeof(seq_confs) / 2, sizeof(split_confs) / 3, chars, output_dev,
    output_count_dev);
  hipDeviceSynchronize();
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  hipMemcpy(&output_count, output_count_dev, sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(output_buf, output_dev, output_count, hipMemcpyDeviceToHost);
  double secs = diff.tv_sec + diff.tv_usec / 1000000.0;
  printf("%.2f MB/s, %d output tokens, random output byte: %d\n", (chars * NUM_THREADS) / 1000000.0 / secs, output_count,
    output_count == 0 ? 0 : output_buf[rand() % output_count]);
  return 0;
}
