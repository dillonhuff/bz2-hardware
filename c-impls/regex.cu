#include "hip/hip_runtime.h"
#include <stdint.h>
#include <sys/time.h>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

using namespace std;

#define NUM_SMS 110
// must be power of two
#define BLOCK_SIZE 256
// must be power of two
#define NUM_THREADS_PER_SM 2048
#define NUM_BLOCKS_PER_SM (NUM_THREADS_PER_SM / BLOCK_SIZE)
#define NUM_BLOCKS (NUM_SMS * NUM_BLOCKS_PER_SM)
#define NUM_THREADS (NUM_THREADS_PER_SM * NUM_SMS)

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define MAX(x, y) (((x) > (y)) ? (x) : (y))

typedef uint8_t uint1_t;

__global__ void run(uint8_t *input_full, uint32_t input_count, uint32_t *output_full, uint32_t *output_count) {
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint8_t *input_buf = input_full + index * input_count;
  uint32_t *output_buf = output_full + index * input_count;

  uint32_t input_idx = 0;
  uint32_t output_buf_idx = 0;

  uint1_t s12 = 0;
  uint1_t s13 = 0;
  uint1_t s10 = 0;
  uint1_t s16 = 0;
  uint1_t s17 = 0;
  uint1_t s11 = 0;
  uint1_t s20 = 0;
  uint1_t s21 = 0;

  #define BUF_SIZE 256
  uint8_t input[BUF_SIZE];
  uint32_t output[BUF_SIZE];
  for (uint32_t ii = input_idx; ii < input_count; ii += BUF_SIZE) {
    for (uint32_t i = ii; i < MIN(input_count, ii + BUF_SIZE); i++) {
      input[i - ii] = input_buf[i];
    }
    uint32_t output_idx = 0;
    for (uint32_t i = 0; i < MIN(input_count - ii, BUF_SIZE); i++) {
      uint1_t o12 = (0 || (input[i] >= '0' && input[i] <= '9') || (input[i] >= 'a' && input[i] <= 'z') || input[i] == '-' || input[i] == '+') && s12;
      s12 = 1;
      uint1_t o13 = (0 || (input[i] >= '0' && input[i] <= '9') || (input[i] >= 'a' && input[i] <= 'z') || input[i] == '-' || input[i] == '+') && s13;
      uint1_t o14 = o13 || o12;
      s13 = o14;
      uint1_t o15 = o14;
      uint1_t o10 = (0 || input[i] == '@') && s10;
      s10 = o15;
      uint1_t o16 = (0 || (input[i] >= '0' && input[i] <= '9') || (input[i] >= 'a' && input[i] <= 'z') || input[i] == '-') && s16;
      s16 = o10;
      uint1_t o17 = (0 || (input[i] >= '0' && input[i] <= '9') || (input[i] >= 'a' && input[i] <= 'z') || input[i] == '-') && s17;
      uint1_t o18 = o17 || o16;
      s17 = o18;
      uint1_t o19 = o18;
      uint1_t o11 = (0 || input[i] == '.') && s11;
      s11 = o19;
      uint1_t o20 = (0 || (input[i] >= '0' && input[i] <= '9') || (input[i] >= 'a' && input[i] <= 'z') || input[i] == '-') && s20;
      s20 = o11;
      uint1_t o21 = (0 || (input[i] >= '0' && input[i] <= '9') || (input[i] >= 'a' && input[i] <= 'z') || input[i] == '-') && s21;
      uint1_t o22 = o21 || o20;
      s21 = o22;
      uint1_t o23 = o22;
      uint1_t o24 = o23;
      uint1_t o25 = o24;
      uint1_t o26 = o25;
      uint1_t o27 = o26;
      if (o27) {
        output[output_idx++] = i;
      }
    }
    for (uint32_t i = 0; i < output_idx; i++) {
      output_buf[output_buf_idx + i] = output[i];
    }
    output_buf_idx += output_idx;
  }
  *(output_count + index) = output_buf_idx;
}

int main(int argc, char **argv) {
  uint32_t GLOBAL_CHARS = 10000000;
  uint32_t CHARS = atoi(argv[1]);

  // extracts "ad_id" and "ad_type"
  uint8_t seq_confs[] = {1, 34, 2, 97, 3, 100, 4, 95, 5, 105, 6, 100, 200, 34, 8, 121, 9, 112, 10, 101, 200, 34};
  uint8_t split_confs[] = {4, 7, 116};

  ifstream infile("kafka-json.txt");
  string line;

  uint32_t conf_size = sizeof(seq_confs) + sizeof(split_confs);
  uint32_t input_buf_size = conf_size + GLOBAL_CHARS;
  uint8_t *input_buf = new uint8_t[input_buf_size];
  uint32_t global_chars = 0;
  uint64_t chars = 0;
  memcpy(input_buf + global_chars, seq_confs, sizeof(seq_confs));
  global_chars += sizeof(seq_confs);
  memcpy(input_buf + global_chars, split_confs, sizeof(split_confs));
  global_chars += sizeof(split_confs);
  while (getline(infile, line)) {
    if (chars == 0 && global_chars + line.length() > conf_size + CHARS) {
      chars = global_chars;
    }
    if (global_chars + line.length() > input_buf_size) {
      break;
    }
    memcpy(input_buf + global_chars, line.c_str(), line.length());
    global_chars += line.length();
  }

  chars = chars / 4 * 4;
  uint8_t *combined_input = new uint8_t[chars * NUM_THREADS];
  for (uint64_t i = 0; i < NUM_THREADS; i++) {
    memcpy(combined_input + i * chars, input_buf + i * 10, chars);
  }

  uint32_t *output_buf = new uint32_t[chars];
  uint32_t output_count;

  uint8_t *input_dev;
  uint32_t *output_dev;
  uint32_t *output_count_dev;
  hipSetDevice(0);
  assert(hipMalloc((void **) &output_dev, chars * sizeof(uint32_t) * NUM_THREADS) == hipSuccess);
  assert(hipMalloc((void **) &input_dev, chars * NUM_THREADS) == hipSuccess);
  assert(hipMalloc((void **) &output_count_dev, sizeof(uint32_t) * NUM_THREADS) == hipSuccess);

  hipMemcpy(input_dev, combined_input, chars * NUM_THREADS, hipMemcpyHostToDevice);

  struct timeval start, end, diff;
  gettimeofday(&start, 0);
  run<<<NUM_BLOCKS, BLOCK_SIZE>>>(input_dev, chars, output_dev, output_count_dev);
  hipDeviceSynchronize();
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  hipMemcpy(&output_count, output_count_dev, sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(output_buf, output_dev, output_count * sizeof(uint32_t), hipMemcpyDeviceToHost);
  double secs = diff.tv_sec + diff.tv_usec / 1000000.0;
  printf("%.2f MB/s, %d input tokens, %d output tokens, random output byte: %d\n",
    (chars * NUM_THREADS) / 1000000.0 / secs, (int) chars, output_count,
    output_count == 0 ? 0 : output_buf[rand() % output_count]);
  return 0;
}
