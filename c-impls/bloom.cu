#include "hip/hip_runtime.h"
#include <stdint.h>
#include <sys/time.h>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

using namespace std;

#define NUM_SMS 110
// must be power of two
#define BLOCK_SIZE 256
// must be power of two
#define NUM_THREADS_PER_SM 2048
#define NUM_BLOCKS_PER_SM (NUM_THREADS_PER_SM / BLOCK_SIZE)
#define NUM_BLOCKS (NUM_SMS * NUM_BLOCKS_PER_SM)
#define NUM_THREADS (NUM_THREADS_PER_SM * NUM_SMS)

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define MAX(x, y) (((x) > (y)) ? (x) : (y))

#define NUM_HASHES 8
#define NUM_ITEMS 3072
#define NUM_BLOOM_BYTES 4096
#define ITEM_BYTES 100

typedef uint8_t uint1_t;

__global__ void run(uint8_t *input_full, uint32_t input_count, uint8_t *output_full, uint32_t *output_count) {
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint8_t *input_buf = input_full + index * input_count;
  uint8_t *output_buf = output_full + index * input_count;

  uint32_t input_idx = 0;
  uint32_t output_buf_idx = 0;

  uint32_t hash_seeds[NUM_HASHES] = {0, 1, 2, 3, 4, 5, 6, 7};
  uint32_t hashes[NUM_HASHES] = {0, 1, 2, 3, 4, 5, 6, 7};
  uint8_t bloom[NUM_BLOOM_BYTES] = {0};
  uint32_t byte_counter = 0;
  uint32_t item_counter = 0;

  #define BUF_SIZE 256
  uint8_t input[BUF_SIZE];
  for (uint32_t ii = input_idx; ii < input_count; ii += BUF_SIZE) {
    for (uint32_t i = ii; i < MIN(input_count, ii + BUF_SIZE); i++) {
      input[i - ii] = input_buf[i];
    }
    for (uint32_t i = 0; i < MIN(input_count - ii, BUF_SIZE); i++) {
      for (uint32_t j = 0; j < NUM_HASHES; j++) {
        hashes[j] += input[i];
        hashes[j] += hashes[j] << 10;
        hashes[j] ^= hashes[j] >> 6;
      }
      byte_counter++;
      if (byte_counter == ITEM_BYTES) {
        for (uint32_t j = 0; j < NUM_HASHES; j++) {
          hashes[j] += hashes[j] << 3;
          hashes[j] ^= hashes[j] >> 11;
          hashes[j] += hashes[j] << 15;
          uint32_t cell = (hashes[j] >> 3) & (NUM_BLOOM_BYTES - 1);
          uint32_t bit = hashes[j] & 7;
          bloom[cell] |= 1 << bit;
          hashes[j] = hash_seeds[j];
        }
        byte_counter = 0;
        item_counter++;
      }
      if (item_counter == NUM_ITEMS) {
        for (uint32_t j = 0; j < NUM_BLOOM_BYTES; j++) {
          output_buf[output_buf_idx++] = bloom[j];
          bloom[j] = 0;
        }
        item_counter = 0;
      }
    }
  }
  *(output_count + index) = output_buf_idx;
}

int main(int argc, char **argv) {
  uint32_t GLOBAL_CHARS = 10000000;
  uint32_t CHARS = atoi(argv[1]);

  // extracts "ad_id" and "ad_type"
  uint8_t seq_confs[] = {1, 34, 2, 97, 3, 100, 4, 95, 5, 105, 6, 100, 200, 34, 8, 121, 9, 112, 10, 101, 200, 34};
  uint8_t split_confs[] = {4, 7, 116};

  ifstream infile("kafka-json.txt");
  string line;

  uint32_t conf_size = sizeof(seq_confs) + sizeof(split_confs);
  uint32_t input_buf_size = conf_size + GLOBAL_CHARS;
  uint8_t *input_buf = new uint8_t[input_buf_size];
  uint32_t global_chars = 0;
  uint64_t chars = 0;
  memcpy(input_buf + global_chars, seq_confs, sizeof(seq_confs));
  global_chars += sizeof(seq_confs);
  memcpy(input_buf + global_chars, split_confs, sizeof(split_confs));
  global_chars += sizeof(split_confs);
  while (getline(infile, line)) {
    if (chars == 0 && global_chars + line.length() > conf_size + CHARS) {
      chars = global_chars;
    }
    if (global_chars + line.length() > input_buf_size) {
      break;
    }
    memcpy(input_buf + global_chars, line.c_str(), line.length());
    global_chars += line.length();
  }

  chars = chars / 4 * 4;
  uint8_t *combined_input = new uint8_t[chars * NUM_THREADS];
  for (uint64_t i = 0; i < NUM_THREADS; i++) {
    memcpy(combined_input + i * chars, input_buf + i * 10, chars);
  }

  uint8_t *output_buf = new uint8_t[chars];
  uint32_t output_count;

  uint8_t *input_dev;
  uint8_t *output_dev;
  uint32_t *output_count_dev;
  hipSetDevice(0);
  assert(hipMalloc((void **) &output_dev, chars * NUM_THREADS) == hipSuccess);
  assert(hipMalloc((void **) &input_dev, chars * NUM_THREADS) == hipSuccess);
  assert(hipMalloc((void **) &output_count_dev, sizeof(uint32_t) * NUM_THREADS) == hipSuccess);

  hipMemcpy(input_dev, combined_input, chars * NUM_THREADS, hipMemcpyHostToDevice);

  struct timeval start, end, diff;
  gettimeofday(&start, 0);
  run<<<NUM_BLOCKS, BLOCK_SIZE>>>(input_dev, chars, output_dev, output_count_dev);
  hipDeviceSynchronize();
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  hipMemcpy(&output_count, output_count_dev, sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(output_buf, output_dev, output_count, hipMemcpyDeviceToHost);
  double secs = diff.tv_sec + diff.tv_usec / 1000000.0;
  printf("%.2f MB/s, %d input tokens, %d output tokens, random output byte: %d\n",
    (chars * NUM_THREADS) / 1000000.0 / secs, (int) chars, output_count,
    output_count == 0 ? 0 : output_buf[rand() % output_count]);
  return 0;
}
