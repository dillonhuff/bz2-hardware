#include "hip/hip_runtime.h"
#include <stdint.h>
#include <sys/time.h>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

using namespace std;

#define NUM_SMS 110
// must be power of two
#define BLOCK_SIZE 256
// must be power of two
#define NUM_THREADS_PER_SM 2048
#define NUM_BLOCKS_PER_SM (NUM_THREADS_PER_SM / BLOCK_SIZE)
#define NUM_BLOCKS (NUM_SMS * NUM_BLOCKS_PER_SM)
#define NUM_THREADS (NUM_THREADS_PER_SM * NUM_SMS)

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define MAX(x, y) (((x) > (y)) ? (x) : (y))

#define BATCH_SIZE 4
#define LG_BATCH_SIZE 2
#define LG_BATCH_SIZE_PLUS_ONE 3
#define WORD_SIZE 32
#define LG_WORD_SIZE 5
#define NUM_WIDTHS 16
#define LG_NUM_WIDTHS 4

typedef uint8_t uint1_t;

typedef struct {
  uint1_t fixed_cheaper;
  uint8_t cost; // may not be large enough at larger batch size
} cost_info;

__device__ uint8_t bit_length(uint32_t word) {
  return word == 0 ? 1 : WORD_SIZE - __clz(word);
}

__device__ uint32_t bit_select(uint32_t word, uint32_t upper, uint32_t lower) {
  uint32_t num_bits = upper - lower + 1;
  return (word >> lower) & ((1L << num_bits) - 1);
}

__device__ cost_info compute_cost(uint8_t width, uint8_t bit_count[3]) {
  uint8_t num_exceptions = BATCH_SIZE - bit_count[0];
  uint8_t fixed_cost = LG_WORD_SIZE + num_exceptions * bit_count[1];
  uint8_t varint_cost = bit_count[2]; // will be 0 if there are no exceptions
  uint8_t common_exception_cost = (num_exceptions > 0 ? 1 : 0) + num_exceptions * MAX(LG_BATCH_SIZE, 1);
  return (cost_info) {fixed_cost <= varint_cost, width * bit_count[0] +
    common_exception_cost + (fixed_cost <= varint_cost ? fixed_cost : varint_cost)};
}

__global__ void run(uint32_t *input_full, uint32_t input_count, uint32_t *output_full, uint32_t *output_count) {
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t *input_buf = input_full + index * input_count;
  uint32_t *output_buf = output_full + index * 2 * input_count;

  uint32_t input_idx = 0;
  uint32_t output_buf_idx = 0;
  uint32_t out_buf = 0;
  uint8_t out_buf_bits = 0;
  uint8_t bits_to_varint_len[33];
  uint8_t bit_widths[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 16, 20, 32};
  uint8_t bit_counts[NUM_WIDTHS][3]; // uint8_t may not be large enough if we have a larger batch size
  uint8_t words_consumed = 0;
  uint32_t buffer[BATCH_SIZE];

  for (int i = 1; i < 33; i++) {
    bits_to_varint_len[i] = (i + 7 - 1) / 7 * 8;
  }
  for (int i = 0; i < NUM_WIDTHS; i++) {
    bit_counts[i][0] = 0;
    bit_counts[i][1] = 0;
    bit_counts[i][2] = 0;
  }

  #define BUF_SIZE 256
  uint32_t input[BUF_SIZE];
  uint32_t output[BUF_SIZE * 2];
  for (uint32_t ii = input_idx; ii < input_count; ii += BUF_SIZE) {
    for (uint32_t i = ii; i < MIN(input_count, ii + BUF_SIZE); i++) {
      input[i - ii] = input_buf[i];
    }
    uint32_t output_idx = 0;
    for (uint32_t i = 0; i < MIN(input_count - ii, BUF_SIZE); i++) {
      #define ADD_TO_OUTPUT(w0, b0) do {\
                uint32_t word = w0;\
                uint8_t num_bits = b0;\
                if (num_bits + out_buf_bits >= 32) {\
                  output[output_idx++] = ((word & ((1L << (32 - out_buf_bits)) - 1)) << out_buf_bits) | out_buf;\
                  out_buf = word >> (32 - out_buf_bits);\
                  out_buf_bits = num_bits + out_buf_bits - 32;\
                } else {\
                  out_buf = (word << out_buf_bits) | out_buf;\
                  out_buf_bits += num_bits;\
                }\
              } while (0)
      // could replace this with the direct computatation instead of table lookup
      #define VARINT_LEN(b0) bits_to_varint_len[b0]

      uint8_t cur_bit_length = bit_length(input[i]);
      for (int j = 0; j < NUM_WIDTHS; j++) {
        if (cur_bit_length <= bit_widths[j]) {
          bit_counts[j][0] += 1;
        } else {
          if (cur_bit_length > bit_counts[j][1]) {
            bit_counts[j][1] = cur_bit_length;
          }
          bit_counts[j][2] += VARINT_LEN(cur_bit_length);
        }
      }
      buffer[words_consumed++] = input[i];
      if (words_consumed == BATCH_SIZE) {
        uint8_t min_width_idx = 0;
        cost_info min_cost = {0, 255}; // assumes that 255 is an unreachable cost value
        for (int j = 0; j < NUM_WIDTHS; j++) {
          cost_info cur_cost = compute_cost(bit_widths[j], bit_counts[j]);
          if (cur_cost.cost < min_cost.cost) {
            min_width_idx = j;
            min_cost = cur_cost;
          }
        }
        ADD_TO_OUTPUT(min_width_idx, MAX(LG_NUM_WIDTHS, 1));
        ADD_TO_OUTPUT(BATCH_SIZE - bit_counts[min_width_idx][0], LG_BATCH_SIZE_PLUS_ONE);
        for (int j = 0; j < BATCH_SIZE; j++) {
          if (bit_length(buffer[j]) <= bit_widths[min_width_idx]) {
            ADD_TO_OUTPUT(buffer[j], bit_widths[min_width_idx]);
          }
        }
        if (bit_counts[min_width_idx][0] < BATCH_SIZE) {
          if (min_cost.fixed_cheaper) {
            // fixed exceptions
            ADD_TO_OUTPUT(0, 1);
            ADD_TO_OUTPUT(bit_counts[min_width_idx][1] - 1, LG_WORD_SIZE);
          } else {
            // varint exceptions
            ADD_TO_OUTPUT(1, 1);
          }
          for (int j = 0; j < BATCH_SIZE; j++) {
            uint8_t word_bit_length = bit_length(buffer[j]);
            if (word_bit_length > bit_widths[min_width_idx]) {
              ADD_TO_OUTPUT(j, MAX(LG_BATCH_SIZE, 1));
              if (min_cost.fixed_cheaper) { // fixed
                ADD_TO_OUTPUT(buffer[j], bit_counts[min_width_idx][1]);
              } else {
                for (int k = 0; k < word_bit_length; k += 7) {
                  if (k + 7 < word_bit_length) {
                    ADD_TO_OUTPUT(bit_select(buffer[j], k + 6, k) | (1 << 7), 8);
                  } else {
                    ADD_TO_OUTPUT(bit_select(buffer[j], word_bit_length - 1, k), 8);
                  }
                }
              }
            }
          }
        }
        for (int j = 0; j < NUM_WIDTHS; j++) {
          bit_counts[j][0] = 0;
          bit_counts[j][1] = 0;
          bit_counts[j][2] = 0;
        }
        words_consumed = 0;
      }
    }
    for (uint32_t i = 0; i < output_idx; i++) {
      output_buf[output_buf_idx + i] = output[i];
    }
    output_buf_idx += output_idx;
  }
  if (out_buf_bits > 0) {
    output_buf[output_buf_idx++] = out_buf;
  }
  *(output_count + index) = output_buf_idx;
}

int main(int argc, char **argv) {
  uint32_t GLOBAL_CHARS = 10000000;
  uint32_t CHARS = atoi(argv[1]);

  // extracts "ad_id" and "ad_type"
  uint8_t seq_confs[] = {1, 34, 2, 97, 3, 100, 4, 95, 5, 105, 6, 100, 200, 34, 8, 121, 9, 112, 10, 101, 200, 34};
  uint8_t split_confs[] = {4, 7, 116};

  ifstream infile("kafka-json.txt");
  string line;

  uint32_t conf_size = sizeof(seq_confs) + sizeof(split_confs);
  uint32_t input_buf_size = conf_size + GLOBAL_CHARS;
  uint8_t *input_buf = new uint8_t[input_buf_size];
  uint32_t global_chars = 0;
  uint64_t chars = 0;
  memcpy(input_buf + global_chars, seq_confs, sizeof(seq_confs));
  global_chars += sizeof(seq_confs);
  memcpy(input_buf + global_chars, split_confs, sizeof(split_confs));
  global_chars += sizeof(split_confs);
  while (getline(infile, line)) {
    if (chars == 0 && global_chars + line.length() > conf_size + CHARS) {
      chars = global_chars;
    }
    if (global_chars + line.length() > input_buf_size) {
      break;
    }
    memcpy(input_buf + global_chars, line.c_str(), line.length());
    global_chars += line.length();
  }

  chars = chars / 4 * 4;
  uint8_t *combined_input = new uint8_t[chars * NUM_THREADS];
  for (uint64_t i = 0; i < NUM_THREADS; i++) {
    memcpy(combined_input + i * chars, input_buf + i * 10, chars);
    uint32_t mask = (1L << ((i + 32) % 33)) - 1; // start with full mask at i = 0 to match CPU version
    uint32_t *slice = (uint32_t *)(combined_input + i * chars);
    for (uint32_t j = 0; j < chars / 4; j++) {
      slice[j] = slice[j] & mask;
    }
  }

  uint8_t *output_buf = new uint8_t[chars * 2];
  uint32_t output_count;

  uint8_t *output_dev, *input_dev;
  uint32_t *output_count_dev;
  hipSetDevice(0);
  hipMalloc((void **) &output_dev, chars * NUM_THREADS);
  hipMalloc((void **) &input_dev, chars * NUM_THREADS);
  hipMalloc((void **) &output_count_dev, sizeof(uint32_t) * NUM_THREADS);

  hipMemcpy(input_dev, combined_input, chars * NUM_THREADS, hipMemcpyHostToDevice);

  struct timeval start, end, diff;
  gettimeofday(&start, 0);
  run<<<NUM_BLOCKS, BLOCK_SIZE>>>((uint32_t *)input_dev, chars / 4, (uint32_t *)output_dev, output_count_dev);
  hipDeviceSynchronize();
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  hipMemcpy(&output_count, output_count_dev, sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(output_buf, output_dev, output_count / 4, hipMemcpyDeviceToHost);
  double secs = diff.tv_sec + diff.tv_usec / 1000000.0;
  printf("%.2f MB/s, %d input tokens, %d output tokens, random output byte: %d\n",
    (chars * NUM_THREADS) / 1000000.0 / secs, (int) chars / 4, output_count,
    output_count == 0 ? 0 : output_buf[rand() % (output_count * 4)]);
  return 0;
}
